#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <iostream>
#include <ctype.h>

#include <math.h>

#define	CEIL(a,b)		((a+b-1)/b)
#define SWAP(a,b,t)		t=b; b=a; a=t;
#define DATAMB(bytes)			(bytes/1024/1024)
#define DATABW(bytes,timems)	((float)bytes/(timems * 1.024*1024.0*1024.0))
#define PI 3.14159265
typedef unsigned char uch;
typedef unsigned long ul;
typedef unsigned int  ui;

uch *TheImg, *CopyImg;					// Where images are stored in CPU
uch *GPUImg, *GPUCopyImg, *GPUResult;	// Where images are stored in GPU

struct ImgProp{
	int Hpixels;
	int Vpixels;
	uch HeaderInfo[54];
	ul Hbytes;
} ip;

#define	IPHB		ip.Hbytes
#define	IPH			ip.Hpixels
#define	IPV			ip.Vpixels
#define	IMAGESIZE	(IPHB*IPV)
#define	IMAGEPIX	(IPH*IPV)



// Kernel that flips the given image horizontally
// each thread only flips a single pixel (R,G,B)
__global__
void imrotate(uch *ImgDst, uch *ImgSrc, ui Vpixels, ui Hpixels, ui BlkPerRow, ui RowBytes, double cosRot, double sinRot)
{	
	__shared__ uch PixBuffer[3072*16];

	ui ThrPerBlk = blockDim.x;
	ui MYbid = blockIdx.x;
	ui MYtid = threadIdx.x;
	ui MYgtid = ThrPerBlk * MYbid + MYtid;

	ui MYrow = MYbid / BlkPerRow;
	ui MYcol = MYgtid - MYrow*BlkPerRow*ThrPerBlk;
	if (MYcol >= Hpixels) return;			// col out of range
	ui MYsrcOffset = MYrow * RowBytes;
	ui MYsrcIndex = MYsrcOffset + 3 * MYcol;

	////////////// find destination index	
	int c, h, v, X, Y, NewCol, NewRow;
	double newX, newY, H, V, Diagonal, ScaleFactor;

	c=MYcol;  		h=Hpixels/2;   v=Vpixels/2;	// integer div
	X=(double)c-(double)h;
	Y=(double)v-(double)MYrow;
	
	// pixel rotation matrix
	newX=cosRot*X-sinRot*Y;
	newY=sinRot*X+cosRot*Y;
	
	// Scale to fit everything in the image box
	H=(double)Hpixels;
	V=(double)Vpixels;
	Diagonal=sqrt(H*H+V*V);
	ScaleFactor=(Hpixels>Vpixels) ? V/Diagonal : H/Diagonal;
	newX=newX*ScaleFactor;
	newY=newY*ScaleFactor;
	
	// convert back from Cartesian to image coordinates
	NewCol=((int) newX+h);
	NewRow=v-(int)newY;
	ui MYdstOffset = NewRow*RowBytes;
	ui MYdstIndex = MYdstOffset + 3 * NewCol;
	///////////////	
	ui Mytid3 = MYtid*3;
	PixBuffer[Mytid3] = ImgSrc[MYsrcIndex];
	PixBuffer[Mytid3+1] = ImgSrc[MYsrcIndex+1];
	PixBuffer[Mytid3+2] = ImgSrc[MYsrcIndex+2];
	__syncthreads();

	// swap pixels RGB   @MYcol , @MYmirrorcol
	ImgDst[MYdstIndex] = PixBuffer[Mytid3];
	ImgDst[MYdstIndex + 1] = PixBuffer[Mytid3+1];
	ImgDst[MYdstIndex + 2] = PixBuffer[Mytid3+2];
}






// Read a 24-bit/pixel BMP file into a 1D linear array.
// Allocate memory to store the 1D image and return its pointer.
uch *ReadBMPlin(char* fn)
{
	static uch *Img;
	FILE* f = fopen(fn, "rb");
	if (f == NULL){	printf("\n\n%s NOT FOUND\n\n", fn);	exit(EXIT_FAILURE); }

	uch HeaderInfo[54];
	fread(HeaderInfo, sizeof(uch), 54, f); // read the 54-byte header
	// extract image height and width from header
	int width = *(int*)&HeaderInfo[18];			ip.Hpixels = width;
	int height = *(int*)&HeaderInfo[22];		ip.Vpixels = height;
	int RowBytes = (width * 3 + 3) & (~3);		ip.Hbytes = RowBytes;
	//save header for re-use
	memcpy(ip.HeaderInfo, HeaderInfo,54);
	printf("\n Input File name: %17s  (%d x %d)   File Size=%lu", fn, 
			ip.Hpixels, ip.Vpixels, IMAGESIZE);
	// allocate memory to store the main image (1 Dimensional array)
	Img  = (uch *)malloc(IMAGESIZE);
	if (Img == NULL) return Img;      // Cannot allocate memory
	// read the image from disk
	fread(Img, sizeof(uch), IMAGESIZE, f);
	fclose(f);
	return Img;
}


// Write the 1D linear-memory stored image into file.
void WriteBMPlin(uch *Img, char* fn)
{
	FILE* f = fopen(fn, "wb");
	if (f == NULL){ printf("\n\nFILE CREATION ERROR: %s\n\n", fn); exit(1); }
	//write header
	fwrite(ip.HeaderInfo, sizeof(uch), 54, f);
	//write data
	fwrite(Img, sizeof(uch), IMAGESIZE, f);
	printf("\nOutput File name: %17s  (%u x %u)   File Size=%lu", fn, ip.Hpixels, ip.Vpixels, IMAGESIZE);
	fclose(f);
}


int main(int argc, char **argv)
{
	// char			Flip = 'H';
	float			tmpKernelExcutionTime, totalKernelExecutionTime; // GPU code run times
	hipError_t		cudaStatus, cudaStatus2;
	hipEvent_t		time1, time2;
	char			InputFileName[255], OutputFileName[255], ProgName[255];
	ui				BlkPerRow;
	// ui 			BlkPerRowInt, BlkPerRowInt2;
	ui				ThrPerBlk = 128, NumBlocks;
	// ui 				NB2, NB4, NB8, RowInts;
	ui				RowBytes;
	hipDeviceProp_t	GPUprop;
	ul				SupportedKBlocks, SupportedMBlocks, MaxThrPerBlk;
	// ui				*GPUCopyImg32, *GPUImg32;
	char			SupportedBlocks[100];
	// int				KernelNum=1;
	char			KernelName[255];
	double			RotAngle, deltaAngle;					// rotation angle
	int 			RotIter;
	int 			TotalIters;
	double 			cosRot, sinRot;
	strcpy(ProgName, "imrotateG");
	if(argc!=4){
		printf("\n\nUsage: ./imrotateG infile outfile N");
		return 0;
	}
	strcpy(InputFileName, argv[1]);
	strcpy(OutputFileName, argv[2]);

	// Create CPU memory to store the input and output images
	TheImg = ReadBMPlin(InputFileName); // Read the input image if memory can be allocated
	if (TheImg == NULL){
		printf("Cannot allocate memory for the input image...\n");
		exit(EXIT_FAILURE);
	}
	CopyImg = (uch *)malloc(IMAGESIZE);
	if (CopyImg == NULL){
		free(TheImg);
		printf("Cannot allocate memory for the input image...\n");
		exit(EXIT_FAILURE);
	}

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		exit(EXIT_FAILURE);
	}
	hipGetDeviceProperties(&GPUprop, 0);
	SupportedKBlocks = (ui)GPUprop.maxGridSize[0] * (ui)GPUprop.maxGridSize[1] * (ui)GPUprop.maxGridSize[2] / 1024;
	SupportedMBlocks = SupportedKBlocks / 1024;
	sprintf(SupportedBlocks, "%lu %c", (SupportedMBlocks >= 5) ? SupportedMBlocks : SupportedKBlocks, (SupportedMBlocks >= 5) ? 'M' : 'K');
	MaxThrPerBlk = (ui)GPUprop.maxThreadsPerBlock;

	// Allocate GPU buffer for the input and output images	
	cudaStatus = hipMalloc((void**)&GPUImg, IMAGESIZE);
	cudaStatus2 = hipMalloc((void**)&GPUCopyImg, IMAGESIZE);
	if ((cudaStatus != hipSuccess) || (cudaStatus2 != hipSuccess)){
		fprintf(stderr, "hipMalloc failed! Can't allocate GPU memory");
		exit(EXIT_FAILURE);
	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(GPUImg, TheImg, IMAGESIZE, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy  CPU to GPU  failed!");
		exit(EXIT_FAILURE);
	}
	RowBytes = (IPH * 3 + 3) & (~3);
	RowBytes = (IPH * 3 + 3) & (~3);
	BlkPerRow = CEIL(IPH,ThrPerBlk);
	NumBlocks = IPV*BlkPerRow; 

	printf("\nNum blocks: %d\n", NumBlocks);
	printf("\nThread per block: %d\n", ThrPerBlk);
	TotalIters = atoi(argv[3]);
	if(TotalIters > 30){
		printf("\nN is too large, should be less or equal to 30\n");
	}
	deltaAngle = 2*PI/float(TotalIters);
	printf("\nTotal iterations: %d\n", TotalIters);

	// iteration to find all images

	strcpy(OutputFileName, argv[2]);
	char* token = strtok(OutputFileName, ".");
	char* OutputFirstName = token;
	token = strtok(NULL, ".");
	char* OutputLastName = token;

	for(RotIter=1; RotIter<=TotalIters; RotIter++){
		char outName[128]="";		
		char tmp[10];
		sprintf(tmp, "%d", RotIter);
		strcat(outName, OutputFirstName);
		strcat(outName, tmp);
		strcat(outName, ".");
		strcat(outName, OutputLastName);

		hipEventCreate(&time1);
		hipEventCreate(&time2);
		hipEventRecord(time1, 0); // record time1 in the first iteration

		RotAngle = (double)(RotIter-1)*deltaAngle;
		cosRot = cos(RotAngle);
		sinRot = sin(RotAngle);
		printf("\nRotation angle = %lf\n", RotAngle);
		imrotate <<< NumBlocks, ThrPerBlk >>> (GPUCopyImg, GPUImg, IPV, IPH, BlkPerRow, RowBytes, cosRot, sinRot);

		hipEventRecord(time2, 0); //record time2 in teh last iteration
		hipEventSynchronize(time1);
		hipEventSynchronize(time2);
		hipEventElapsedTime(&tmpKernelExcutionTime, time1, time2);
		totalKernelExecutionTime += tmpKernelExcutionTime;



		strcpy(KernelName, "imrotate : Each thread rotate 1 pixel. Computes everything.\n");
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "\n\nhipDeviceSynchronize returned error code %d after launching the kernel!\n", cudaStatus);
			exit(EXIT_FAILURE);
		}
		GPUResult = GPUCopyImg;
		cudaStatus = hipMemcpy(CopyImg, GPUResult, IMAGESIZE, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy GPU to CPU  failed!");
			exit(EXIT_FAILURE);
		}
		cudaStatus = hipDeviceSynchronize();
			//checkError(cudaGetLastError());	// screen for errors in kernel launches
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "\n Program failed after cudaDeviceSynchronize()!");
			free(TheImg);
			free(CopyImg);
			exit(EXIT_FAILURE);
		}
		WriteBMPlin(CopyImg, outName);		// Write the flipped image back to disk
		memset(CopyImg, 0, IMAGESIZE);
		hipMemset(GPUCopyImg, 0, IMAGESIZE);

	}

	printf("\nTotal Kernel Execution    =%7.2f ms\n", totalKernelExecutionTime);
	
	hipFree(GPUImg);
	hipFree(GPUCopyImg);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		free(TheImg);
		free(CopyImg);
		exit(EXIT_FAILURE);
	}
	free(TheImg);
	free(CopyImg);
	return(EXIT_SUCCESS);
}



